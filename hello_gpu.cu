
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello World from GPU %d!\n",threadIdx.x);
}

int main(void) {
    printf("Hello World from CPU\n");
    helloFromGPU <<<1, 100>>>();
    hipDeviceSynchronize();
    return 0;
}