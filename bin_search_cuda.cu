
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__
void bin_search(int* a, int* l, int* r, int* e, int* searchValue) {
	int idx = threadIdx.x;
	int lm = l[0];
	int rm = r[0];
	int gap = (int)ceil((float)(rm-lm+1)/(float)(256));
	int num_proc = (int)ceil((float)(rm - lm + 1)/(float)gap);
	int currl = idx*gap + lm;
	if(currl > rm) return;
	int currr = min((idx+1)*gap + lm,rm+1) - 1;
	if(searchValue[0] >= a[currl] && searchValue[0] <= a[currr]) {
		l[0] = currl;
		r[0] = currr;
	}
}

int main(int argc, char* argv[]) {
	int n;
	scanf("%d",&n);
	int *a;
	int *searchValue;
	hipMallocManaged(&a, n*sizeof(int));
	hipMallocManaged(&searchValue, sizeof(int));
	scanf("%d",&searchValue[0]);
	for(int i=0;i<n;i++) scanf("%d",&a[i]);
	int *l, *r;
	int *e;
	hipMallocManaged(&l, sizeof(int));
	hipMallocManaged(&r, sizeof(int));
	hipMallocManaged(&e, sizeof(int));
	l[0] = 0; r[0] = (n-1);
	while(l[0] < r[0]) {
		bin_search<<<1,256>>>(a,l,r,e,searchValue);
		hipDeviceSynchronize();
	}
	printf("%d\n",l[0]);
	hipFree(a);
	hipFree(l);
	hipFree(r);
	hipFree(e);
	hipFree(searchValue);
	return 0;
}