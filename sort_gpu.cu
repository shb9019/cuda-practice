
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void count_sort(int n, int* a, int* b, int* c) {
    int i = threadIdx.x;
    int iinc = blockDim.x;

    int j = 0;
    int jinc = gridDim.x;

    for(; i < n; i += iinc) {
        for(;j<n;j+=1) {
            if((a[i] > a[j]) || (a[i] == a[j] && i > j)) {
                c[i]++;
            }
        }
    }
}

int main(void) {
    int n = 2048;
    int *a, *b, *c;
    hipMallocManaged(&a, n*sizeof(int));
    hipMallocManaged(&b, n*sizeof(int));
    hipMallocManaged(&c, n*sizeof(int));

    for(int i=0;i<n;i++) a[i] = (i+1);
    for(int i=0;i<n;i++) c[i] = 0;

    count_sort<<<1,256>>>(n,a,b,c);
    hipDeviceSynchronize();

    bool flag = true;
    for(int i=0;i<n;i++) {
        printf("%d ",c[i]);
        if(c[i] != i) flag = false;
    }
    if(flag) printf("\nTrue \n");
    else printf("\nFalse \n");

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}