#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void bin_search(int* a, int* l, int* r, int* e, int* searchValue) {
	int lm = l[0];
	int rm = r[0];
	int mm = ((lm+rm)/2);
	if(a[mm] >= searchValue[0]) rm = mm;
	else lm = mm+1;
	l[0] = lm;
	r[0] = rm;
}

int main(int argc, char* argv[]) {
	int n;
	scanf("%d",&n);
	int *a;
	int *searchValue;
	hipMallocManaged(&a, n*sizeof(int));
	hipMallocManaged(&searchValue, sizeof(int));
	scanf("%d",&searchValue[0]);
	for(int i=0;i<n;i++) scanf("%d",&a[i]);
	int *l, *r;
	int *e;
	hipMallocManaged(&l, sizeof(int));
	hipMallocManaged(&r, sizeof(int));
	hipMallocManaged(&e, sizeof(int));

	l[0] = 0; r[0] = (n-1);
	clock_t t; 
	t = clock();
	while(l[0] < r[0]) {
		bin_search<<<1,1>>>(a,l,r,e,searchValue);
		hipDeviceSynchronize();
	}
	t = clock() - t; 
	printf("%d",(int)t);
	hipFree(a);
	hipFree(l);
	hipFree(r);
	hipFree(e);
	hipFree(searchValue);
	return 0;
}